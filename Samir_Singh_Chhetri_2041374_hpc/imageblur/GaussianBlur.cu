#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"


//compile with c++ lodepng file

//nvcc GaussianBlur.cu lodepng.cpp

__global__ void blurKernel(unsigned char * gpu_imageOuput, unsigned char * gpu_imageInput,unsigned int width,unsigned int height) 
{ 
	        int r = 0;
		int g = 0;
		int b = 0;
		int t = 0;
		int x,y;
		int count = 0;

		int idx = blockDim.x * blockIdx.x + threadIdx.x;
		int pixel = idx*4;

		for(x = (pixel - 4); x <=  (pixel + 4); x+=4){
			// Checking conditions so pixel is available at x
			if ((x > 0) && x < (height * width * 4) && ((x-4)/(4*width) == pixel/(4*width))){
				for(y = (x - (4 * width)); y <=  (x + (4 * width)); y+=(4*width)){
					if(y > 0 && y < (height * width * 4)){
						r += gpu_imageInput[y];
						g += gpu_imageInput[1+y];
						b += gpu_imageInput[2+y]; 
						count++;
					}
				}
			}
		}
		
		t = gpu_imageInput[3+pixel];

		gpu_imageOuput[pixel] = r / count;
		gpu_imageOuput[1+pixel] = g / count;
		gpu_imageOuput[2+pixel] = b / count;
		gpu_imageOuput[3+pixel] = t;
}

int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main(int argc, char **argv){

struct timespec start, finish;
    	long long int time_elapsed;
    	 //Start Timer
    	clock_gettime(CLOCK_MONOTONIC, &start);
    	
	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "hck.png";
	const char* newFileName = "Output.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	blurKernel<<<height, width>>>(d_out, d_in, width,height);

	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}

	hipFree(d_in);
	hipFree(d_out);
	
	clock_gettime(CLOCK_MONOTONIC, &finish);
        time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,(time_elapsed/1.0e9));
	
	

	return 0;
}
