#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables

// Compile this program with --->  nvcc -o PasswordCrack PasswordCrack.cu 



//This function encrypt the plain provided values using CudaCrypt method and compares the result with encrypted password and finds the password


__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}
__device__ int is_match(char* attempt){
       char password[]="sc55";
       
       char *a=attempt;
       
       char *p=CudaCrypt(password);
       //printf("Encrypted Password: %s\n",a);
       // printf("Plain Password: %s\n",p);
	while (*a == *p){
	//printf("possible Plain Passwords: %s\n",a);
	if (*a == '\0')
	{
	printf("Encrypted Password: %s\n",attempt);
	printf("found password: %s\n",password);
	break;
	}
	a++;
	p++;
	}
	return 0;
}

__global__ void crack(char * alphabet, char * numbers){	
	char genRawPass[4];

	genRawPass[0] = alphabet[blockIdx.x];
	genRawPass[1] = alphabet[blockIdx.y];

	genRawPass[2] = numbers[threadIdx.x];
	genRawPass[3] = numbers[threadIdx.y];


	char *generated=CudaCrypt(genRawPass);
	//firstLetter - 'a' - 'z' (26 characters)
       //secondLetter - 'a' - 'z' (26 characters)
       //firstNum - '0' - '9' (10 characters)
       //secondNum - '0' - '9' (10 characters)
	is_match(generated);


}

int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main(int argc, char ** argv){
	struct timespec start, finish;
    	long long int time_elapsed;
    	 //Start Timer
    	clock_gettime(CLOCK_MONOTONIC, &start);

	char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
	char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};

	char * gpuAlphabet;
	hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
	hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

	char * gpuNumbers;
	hipMalloc( (void**) &gpuNumbers, sizeof(char) * 26); 
	hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);

	crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers);
	hipDeviceSynchronize();
	
	clock_gettime(CLOCK_MONOTONIC, &finish);
        time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,(time_elapsed/1.0e9));
	
return 0;
}
